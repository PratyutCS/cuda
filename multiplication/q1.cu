#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdint.h>
#include <gmp.h>
#include <stdbool.h>
#include <ctype.h>
#include <hip/hip_runtime.h>

// Macro for checking CUDA errors
#define CHECK_CUDA_ERROR(call) { \
  hipError_t err = call; \
  if (err != hipSuccess) { \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(EXIT_FAILURE); \
  } \
}

__global__ void kernel(uint32_t* num1, uint32_t* num2, size_t s1, size_t s2, uint32_t* d_result) {
  for (size_t i = 0; i < s1 + s2; i++) {
      d_result[i] = 0;
  }
  
  for (size_t i = 0; i < s1; i++) {
      uint64_t carry = 0;
      
      for (size_t j = 0; j < s2; j++) {
        size_t pos = i + j;
        uint64_t prod = (uint64_t)num1[i] * (uint64_t)num2[j] + d_result[pos] + carry;
          
        d_result[pos] = (uint32_t)(prod & 0xFFFFFFFF);
          
        carry = prod >> 32;
      }
      
      size_t idx = i + s2;
      while (carry > 0 && idx < (s1+s2)) {
          uint64_t sum = (uint64_t)d_result[idx] + carry;
          d_result[idx] = (uint32_t)(sum & 0xFFFFFFFF);
          carry = sum >> 32;
          idx++;  // Move forward, not backward
      }
  }
}





mpz_t * readFile(const char * fileName, int * size) {
    //printf("[READFILE] inputfileName is: %s\n", fileName);
    FILE * inputFile = fopen(fileName, "r");
  
    if (!inputFile) {
      perror("Error opening file");
      return NULL;
    }
  
    char * line = NULL;
    size_t line_length = 0;
    size_t line_count = 0;
  
    while (getline( & line, & line_length, inputFile) != -1) {
      line_count++;
    }
  
    //printf("[READFILE] line count is: %zd\n", line_count);
    mpz_t * array = (mpz_t * ) malloc(line_count * sizeof(mpz_t));
    rewind(inputFile);
  
    for (size_t i = 0; i < line_count; i++) {
      mpz_init(array[i]);
      if (getline( & line, & line_length, inputFile) == -1) {
        perror("Error reading line");
        fclose(inputFile);
        free(line);
        for (size_t j = 0; j < i; j++) {
          mpz_clear(array[j]);
        }
        free(array);
        return NULL;
      }
      mpz_set_str(array[i], line, 16);
    }
  
    fclose(inputFile);
    free(line);
    * size = line_count;
    return array;
}

char* uint32_array_to_hex(uint32_t* arr, size_t size) {
  if (!arr || size == 0) return NULL;
  
  size_t hex_str_size = (size * 8) + 1;
  char* hex_str = (char*)malloc(hex_str_size);
  if (!hex_str) {
      perror("Memory allocation failed");
      return NULL;
  }
  
  char* ptr = hex_str;
  for (size_t i = 0; i < size; i++) {
      sprintf(ptr, "%08X", arr[size - 1 - i]);
      ptr += 8;
  }
  *ptr = '\0';
  
  return hex_str;
}

// String comparison function (unchanged)
bool compare_strings(char *str1, char *str2) {
  size_t len2 = strlen(str2);
  char *str3 = (char*)malloc(len2 + 1 * sizeof(char));
  if (!str3) {
      fprintf(stderr, "Memory allocation failed.\n");
      exit(EXIT_FAILURE);
  }

  for (size_t i = 0; i < len2; i++) {
      str3[i] = toupper((unsigned char)str2[i]);
  }
  str3[len2] = '\0';

  size_t length1 = strlen(str1);
  size_t length2_actual = strlen(str3);
  
  size_t min_length = (length1 < length2_actual) ? length1 : length2_actual;

  bool flag = true;
  
  for (size_t i = 0; i < min_length; i++) {
    if (str1[length1 - 1 - i] != str3[length2_actual - 1 - i]) {
      // printf("Difference found: %c from str1 vs %c from str3\n",str1[length1 - 1 - i], str3[length2_actual - 1 - i]);
      FILE *file = fopen("output.txt", "a");
      if (file) {
        fprintf(file, "Difference found: %c from str1 vs %c from str3\n", str1[length1 - 1 - i], str3[length2_actual - 1 - i]);
        fclose(file);
      }
      flag = false;
    }
  }
  free(str3);
  return flag;
}

int main(){
  int line_size;
  mpz_t* fileData = readFile("./input-1000.txt", &line_size);

  long row = 0;
  for (int i = line_size; i >= 1; i = (i + 1) / 2) {
    row++;
    if (i == 1)
      break;
  }

  mpz_t ** array_of_arrays = (mpz_t ** ) malloc(row * sizeof(mpz_t * ));

  uint32_t*** aoa = (uint32_t***) malloc(row * sizeof(uint32_t**));
  size_t** s = (size_t**) malloc(row * sizeof(size_t*));

  int n_line_size = line_size;
  int prev = 0;

  for(int i=0 ; i<row ; i++){
    printf("level %d\n",i);
    array_of_arrays[i] = (mpz_t*) malloc(n_line_size * sizeof(mpz_t));

    for(int j=0; j<n_line_size; j++){
      mpz_init(array_of_arrays[i][j]);
    }

    if(i == 0){
      printf("0 exec\n");
        ////CPU////
      for(int j=0 ; j<n_line_size ; j++){
        mpz_set(array_of_arrays[i][j], fileData[j]);
      }

        ////GPU////

      aoa[i] = (uint32_t**) malloc(n_line_size * sizeof(uint32_t*));
      s[i] = (size_t*) malloc(n_line_size * sizeof(size_t));

      for (int k = 0; k < n_line_size; k++) {
        size_t s1 = ((mpz_sizeinbase(fileData[k], 2) + 31) / 32);
        s[0][k] = s1;
        aoa[0][k] = (uint32_t*) malloc(s1 * sizeof(uint32_t));
        mpz_t temp;
        mpz_init_set(temp, fileData[k]);
        for (size_t j = 0; j < s1; j++) {
          aoa[0][k][j] = (uint32_t) mpz_get_ui(temp);
          mpz_fdiv_q_2exp(temp, temp, 32);
        }
        mpz_clear(temp);
      }

      for(int j=0 ; j<n_line_size ; j++){
        char * self_result = uint32_array_to_hex(aoa[i][j],s[i][j]);
        char * gmp_result = mpz_get_str(NULL, 16, array_of_arrays[i][j]);

        if (compare_strings(self_result, gmp_result)) {
          printf("The strings match (when compared from the end)! for %d\n",j);
        } 
        else {
          printf("The strings do not match (when compared from the end)! for %d\n",j);
          FILE *file = fopen("output.txt", "a");
          if (file) {
            fprintf(file, "SELF: %s\n", self_result);
            fprintf(file, "GMP_: %s\n", gmp_result);
            fprintf(file, "\n");
            fclose(file);
          }
        }
      }
      
      prev = n_line_size;
      n_line_size = (n_line_size+1)/2;
      continue;
    }

    printf("mul execution\n");

      ////CPU////

    for(int j=0 ; j<n_line_size ; j++){
      if((j*2+1) < prev){
        mpz_mul(array_of_arrays[i][j], array_of_arrays[i-1][(j*2)], array_of_arrays[i-1][(j*2)+1]);
      } else {
        mpz_set(array_of_arrays[i][j], array_of_arrays[i-1][(j*2)]);
      }
    }

    ////GPU////

    aoa[i] = (uint32_t**) malloc(n_line_size * sizeof(uint32_t*));
    s[i] = (size_t*) malloc(n_line_size * sizeof(size_t));

    for(int j=0 ; j<n_line_size ; j++){
      if((j*2+1) >= prev){
        size_t s1 = s[i-1][j*2];
        aoa[i][j] = (uint32_t*)malloc(s1 * sizeof(uint32_t));
        memcpy(aoa[i][j], aoa[i-1][j*2], s1 * sizeof(uint32_t));
        s[i][j] = s1;
        printf("copied down below for %d\n",j);
        continue;
      } 
      else {
        size_t s1 = s[i-1][j*2];
        size_t s2 = s[i-1][((j*2)+1)];
        aoa[i][j] = (uint32_t*) malloc((s1+s2) * sizeof(uint32_t));
        s[i][j] = s1 + s2;
        uint32_t *d_num1, *d_num2, *d_result;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_num1, s1 * sizeof(uint32_t)));
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_num2, s2 * sizeof(uint32_t)));
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_result, (s1+s2) * sizeof(uint32_t)));
    
        CHECK_CUDA_ERROR(hipMemcpy(d_num1, aoa[i-1][(j*2)], s1 * sizeof(uint32_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_num2, aoa[i-1][(j*2)+1], s2 * sizeof(uint32_t), hipMemcpyHostToDevice));
                
        // Initialize result array to zeros
        CHECK_CUDA_ERROR(hipMemset(d_result, 0, (s1+s2) * sizeof(uint32_t)));
    
        kernel<<<1,1>>>(d_num1, d_num2, s1, s2, d_result);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
        CHECK_CUDA_ERROR(hipGetLastError());
    
        CHECK_CUDA_ERROR(hipMemcpy(aoa[i][j], d_result, (s1+s2) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    
        CHECK_CUDA_ERROR(hipFree(d_num1));
        CHECK_CUDA_ERROR(hipFree(d_num2));
        CHECK_CUDA_ERROR(hipFree(d_result));
      }
    }

    for(int j=0 ; j<n_line_size ; j++){
      char * self_result = uint32_array_to_hex(aoa[i][j],s[i][j]);
      char * gmp_result = mpz_get_str(NULL, 16, array_of_arrays[i][j]);

      if (compare_strings(self_result, gmp_result)) {
        printf("The strings match (when compared from the end)! for %d\n",j);
      } 
      else {
        printf("The strings do not match (when compared from the end)! for %d\n",j);
        FILE *file = fopen("output.txt", "a");
        if (file) {
          fprintf(file, "SELF: %s\n", self_result);
          fprintf(file, "GMP_: %s\n", gmp_result);
          fprintf(file, "\n");
          fclose(file);
        }
      }
    }

    prev = n_line_size;
    n_line_size = (n_line_size+1)/2;
  }
  // mpz_out_str(stdout, 16, array_of_arrays[row-1][0]);
  printf("\n");
}